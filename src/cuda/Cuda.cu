#include <cuda/hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace PLVS
{

namespace cuda
{

void deviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

}

}
