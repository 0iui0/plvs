#include <cuda/hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace PLVS2
{

namespace cuda
{

void deviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

}

}
